#include "hip/hip_runtime.h"
// This code is a modification of L2 cache benchmark from
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking":
// https://arxiv.org/pdf/1804.06826.pdf

// This benchmark measures the maximum read bandwidth of L2 cache for 32f
// Compile this file using the following command to disable L1 cache:
//    nvcc -Xptxas -dlcm=cg -Xptxas -dscm=wt l2_bw.cu

#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "../../../hw_def/hw_def.h"

#define REPEAT_TIMES 256

/*
L2 cache is warmed up by loading posArray and adding sink
Start timing after warming up
Load posArray and add sink to generate read traffic
Repeat the previous step while offsetting posArray by one each iteration
Stop timing and store data
*/

__global__ void l2_bw(uint64_t *startClk, uint64_t *stopClk, float *dsink,
                      float *posArray, unsigned ARRAY_SIZE) {
  // block and thread index
  uint32_t tid = threadIdx.x;
  uint32_t bid = blockIdx.x;
  uint32_t uid = bid * blockDim.x + tid;

  // a register to avoid compiler optimization
  float sink0 = 0;
  float sink1 = 0;
  float sink2 = 0;
  float sink3 = 0;

  // warm up l2 cache
  for (uint32_t i = uid; i < ARRAY_SIZE; i += blockDim.x * gridDim.x) {
    float *ptr = posArray + i;
    // every warp loads all data in l2 cache
    // use cg modifier to cache the load in L2 and bypass L1
    asm volatile("{\t\n"
                 ".reg .f32 data;\n\t"
                 "ld.global.cg.f32 data, [%1];\n\t"
                 "add.f32 %0, data, %0;\n\t"
                 "}"
                 : "+f"(sink0)
                 : "l"(ptr)
                 : "memory");
  }

  asm volatile("bar.sync 0;");

  // start timing
  uint64_t start = 0;
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");

  // load data from l2 cache and accumulate,
  for (uint32_t i = 0; i < REPEAT_TIMES; i++) {
    float *ptr = posArray + (((i * warpSize * 4) + uid * 4) % ARRAY_SIZE);
    asm volatile("{\t\n"
                 ".reg .f32 data<4>;\n\t"
                 "ld.global.cg.v4.f32 {data0,data1,data2,data3}, [%4];\n\t"
                 "add.f32 %0, data0, %0;\n\t"
                 "add.f32 %1, data1, %1;\n\t"
                 "add.f32 %2, data2, %2;\n\t"
                 "add.f32 %3, data3, %3;\n\t"
                 "}"
                 : "+f"(sink0), "+f"(sink1), "+f"(sink2), "+f"(sink3)
                 : "l"(ptr)
                 : "memory");
  }
  asm volatile("bar.sync 0;");

  // stop timing
  uint64_t stop = 0;
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");

  // store the result
  startClk[uid] = start;
  stopClk[uid] = stop;
  dsink[uid] = sink0 + sink1 + sink2 + sink3;
}

int main() {

  intilizeDeviceProp(0);

  unsigned ARRAY_SIZE = TOTAL_THREADS * 4 + REPEAT_TIMES * WARP_SIZE * 4;
  // Array size must not exceed L2 size
  assert(ARRAY_SIZE * sizeof(float) < L2_SIZE);

  uint64_t *startClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
  uint64_t *stopClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));

  float *posArray = (float *)malloc(ARRAY_SIZE * sizeof(float));
  float *dsink = (float *)malloc(TOTAL_THREADS * sizeof(float));

  float *posArray_g;
  float *dsink_g;
  uint64_t *startClk_g;
  uint64_t *stopClk_g;

  assert(ARRAY_SIZE < L2_SIZE);

  for (int i = 0; i < ARRAY_SIZE; i++)
    posArray[i] = (float)i;

  gpuErrchk(hipMalloc(&posArray_g, ARRAY_SIZE * sizeof(float)));
  gpuErrchk(hipMalloc(&dsink_g, TOTAL_THREADS * sizeof(float)));
  gpuErrchk(hipMalloc(&startClk_g, TOTAL_THREADS * sizeof(uint64_t)));
  gpuErrchk(hipMalloc(&stopClk_g, TOTAL_THREADS * sizeof(uint64_t)));

  gpuErrchk(hipMemcpy(posArray_g, posArray, ARRAY_SIZE * sizeof(float),
                       hipMemcpyHostToDevice));

  l2_bw<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, dsink_g,
                                           posArray_g, ARRAY_SIZE);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchk(hipMemcpy(startClk, startClk_g, TOTAL_THREADS * sizeof(uint64_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS * sizeof(uint64_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(dsink, dsink_g, TOTAL_THREADS * sizeof(float),
                       hipMemcpyDeviceToHost));

  float bw, BW;
  unsigned long long data =
      (unsigned long long)TOTAL_THREADS * REPEAT_TIMES * sizeof(float) * 4;
  uint64_t total_time = stopClk[0] - startClk[0];
  bw = (float)(data) / ((float)(stopClk[0] - startClk[0]));
  BW = bw * CLK_FREQUENCY * 1000000 / 1024 / 1024 / 1024;
  std::cout << "L2 bandwidth = " << bw << "(byte/clk), " << BW << "(GB/s)\n";
  float max_bw = get_num_channels(MEM_BITWIDTH, DRAM_MODEL) *
                 L2_BANKS_PER_MEM_CHANNEL * L2_BANK_WIDTH_in_BYTE;
  BW = max_bw * CLK_FREQUENCY * 1000000 / 1024 / 1024 / 1024;
  std::cout << "Max Theortical L2 bandwidth = " << max_bw << "(byte/clk), "
            << BW << "(GB/s)\n";
  std::cout << "L2 BW achievable = " << (bw / max_bw) * 100 << "%\n";
  std::cout << "Total Clk number = " << total_time << "\n";

  return 1;
}
