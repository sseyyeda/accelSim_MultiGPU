#include "hip/hip_runtime.h"
/*
This microbenchmark detects L2 write policy
check the nvprof or nvsight for received l2 reads and writes to detect the
policy check the comments below for further details and also see our arvix
paper: https://arxiv.org/pdf/1810.07269.pdf

Compile this file using the following command to disable L1 cache:
    nvcc -Xptxas -dlcm=cg l2_write_policy.cu

to run the program with nvsight
 make nvsight ./l2_write_policy
*/

#include <assert.h>
#include <cstdio>
#include <iostream>
using namespace std;

#include "../../../hw_def/hw_def.h"

#define THREADS_NUM 1   // Launch only one thread
#define ARRAY_SIZE 1024 // size of the array

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Device code

/*
check the nvprof or nvsight to see the L2/DRAM reads and write hits
in the below mb, we have 6 reads and 4 writes

1. Check the write allocation policy
 we have three policies: write no-allocate vs write-allocate fetch-on-write vs
vs write-allocate sub-sector write?? if only two write hits (C[i] and A[i] at
lines 3&4) ==> then write no-allocate, else if three write hits (C[i+1], C[i]
and A[i] at lines 2&3&4) then it is write-allocate. if one read miss (A[i] at
line1) and 5 reads hits ==> then fetch-on-write, as the miss at line1 will fetch
the whole sector, and C[i] and line 3 is hit else if two read misses (A[i] at
lines 1 and C[i] at line 3) ==> then sub-sector write with write bit-mask, the
sector will be fetched on read miss, not write miss

2. check if write-back or write-through
check the DRAM writes, if four writes are received ==> then write-through
if less than four writes ==> then write-back

to run the program with nvsight:
 make nvsight ./l2_write_policy

 stats to look at:
l2 reads: lts__t_sectors_srcunit_tex_op_read.sum
l2 writes: lts__t_sectors_srcunit_tex_op_write.sum
l2 read hits: lts__t_sectors_srcunit_tex_op_read_lookup_hit.sum
l2 write hits: lts__t_sectors_srcunit_tex_op_write_lookup_hit.

The comments below shows a case of write-allocate with sub-sector mask as in
Pascal, Volta, Turing and Ampere HW Results found: Pascal, Volta, Turing and
Ampere: write allocate &  sub-sector write  + write-back
*/

__global__ void write_policy_mb(float *A, float *C) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i == 0) {
    C[i] = A[i];        // write to C[i] is a miss (cache line is missing)
    C[i + 1] = A[i];    // write to C[i+1] is a hit (cache line is found)
    C[i] = C[i] + A[i]; // read of C[i] is a miss (entire sector is missing,
                        // fetch it from memory)
    A[i] =
        C[i] + C[i + 1]; // read C[i] and C[i+1] are hits (entire sector exists)
  }
}

//////////////////////////////////////////////////////
int main(int argc, char *argv[]) {
  intilizeDeviceProp(0);

  BLOCKS_NUM = 1;
  TOTAL_THREADS = THREADS_NUM * BLOCKS_NUM;
  THREADS_PER_SM = THREADS_NUM * BLOCKS_NUM;

  // create 4KB buffers of A&C
  assert(ARRAY_SIZE * sizeof(float) < L2_SIZE);

  float *A = (float *)malloc(ARRAY_SIZE * sizeof(float));
  float *C = (float *)malloc(ARRAY_SIZE * sizeof(float));

  float *A_g;
  float *C_g;

  gpuErrchk(hipMalloc(&A_g, ARRAY_SIZE * sizeof(float)));
  gpuErrchk(hipMalloc(&C_g, ARRAY_SIZE * sizeof(float)));

  for (uint32_t i = 0; i < ARRAY_SIZE; i++)
    A[i] = (float)i;

  gpuErrchk(
      hipMemcpy(A_g, A, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice));

  write_policy_mb<<<1, THREADS_NUM>>>(A_g, C_g);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchk(hipMemcpy(C, C_g, ARRAY_SIZE * sizeof(uint32_t),
                       hipMemcpyDeviceToHost));

  std::cout << "\nThis microbenchmark detects L2 write policy.\n";
  std::cout << "check the nvprof or nvsight for received L2 reads and writes "
               "to detect the policy.\n";
  std::cout << "see the code comments for further details\n";
  std::cout << "to run the program with nvsight: make nvsight ./2\n";
  std::cout << "stats to look at: llts__t_sectors_srcunit_tex_op_read.sum & "
               "lts__t_sectors_srcunit_tex_op_write.sum & "
               "lts__t_sectors_srcunit_tex_op_read_lookup_hit.sum & "
               "lts__t_sectors_srcunit_tex_op_write_lookup_hit.sum \n\n";

  return 1;
}
