#include "hip/hip_runtime.h"

// This benchmark measures the kernel overhead as linear function a + Xb where X
// is the number of launched TBs, a is kernel launch latency and b is TB launch
// latency

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "../../../hw_def/hw_def.h"

#define THREADS_NUM 1024
#define ARRAY_SIZE 4096

__global__ void kernel_lat_1TB(uint32_t *startClk, uint32_t *stopClk,
                               uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_2TB(uint32_t *startClk, uint32_t *stopClk,
                               uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_4TB(uint32_t *startClk, uint32_t *stopClk,
                               uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_8TB(uint32_t *startClk, uint32_t *stopClk,
                               uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_16TB(uint32_t *startClk, uint32_t *stopClk,
                                uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_32TB(uint32_t *startClk, uint32_t *stopClk,
                                uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_64TB(uint32_t *startClk, uint32_t *stopClk,
                                uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_128TB(uint32_t *startClk, uint32_t *stopClk,
                                 uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_256TB(uint32_t *startClk, uint32_t *stopClk,
                                 uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_512TB(uint32_t *startClk, uint32_t *stopClk,
                                 uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_1024TB(uint32_t *startClk, uint32_t *stopClk,
                                  uint64_t *posArray, uint64_t *dsink) {}

__global__ void kernel_lat_2048TB(uint32_t *startClk, uint32_t *stopClk,
                                  uint64_t *posArray, uint64_t *dsink) {}

int main() {
  intilizeDeviceProp(0);

  uint32_t *startClk = (uint32_t *)malloc(THREADS_NUM * sizeof(uint32_t));
  uint32_t *stopClk = (uint32_t *)malloc(THREADS_NUM * sizeof(uint32_t));
  uint64_t *dsink = (uint64_t *)malloc(THREADS_NUM * sizeof(uint64_t));

  uint32_t *startClk_g;
  uint32_t *stopClk_g;
  uint64_t *posArray_g;
  uint64_t *dsink_g;

  gpuErrchk(hipMalloc(&startClk_g, THREADS_NUM * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&stopClk_g, THREADS_NUM * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&posArray_g, ARRAY_SIZE * sizeof(uint64_t)));
  gpuErrchk(hipMalloc(&dsink_g, THREADS_NUM * sizeof(uint64_t)));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  kernel_lat_1TB<<<1, THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
                                     dsink_g);

  gpuErrchk(hipPeekAtLastError());
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  /*
  kernel_lat_2TB<<<2,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g, dsink_g);
  gpuErrchk( hipPeekAtLastError() );

  kernel_lat_4TB<<<4,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g, dsink_g);
  gpuErrchk( hipPeekAtLastError() );

  kernel_lat_8TB<<<8,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g, dsink_g);
  gpuErrchk( hipPeekAtLastError() );

  kernel_lat_16TB<<<16,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );

  kernel_lat_32TB<<<32,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );


  kernel_lat_64TB<<<64,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );

  kernel_lat_128TB<<<128,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );


  kernel_lat_256TB<<<256,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );

  kernel_lat_512TB<<<1024,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );

  kernel_lat_1024TB<<<1024,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );

  kernel_lat_2048TB<<<2048,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g,
  dsink_g); gpuErrchk( hipPeekAtLastError() );

*/

  gpuErrchk(hipMemcpy(startClk, startClk_g, THREADS_NUM * sizeof(uint32_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(stopClk, stopClk_g, THREADS_NUM * sizeof(uint32_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(dsink, dsink_g, THREADS_NUM * sizeof(uint64_t),
                       hipMemcpyDeviceToHost));

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float lat = (milliseconds * 1000 * CLK_FREQUENCY) / 3;
  std::cout << "Kernel Launch Latency = " << lat << " cycles\n";
  std::cout << "The reported latency above can be slightly higher than real. "
               "For accurate evaultion using nvprof event, exmaple: make "
               "events ./kernel_lat\n";

  if (ACCEL_SIM_MODE) {
    std::cout << "\n//Accel_Sim config: \n";
    std::cout << "-gpgpu_kernel_launch_latency  " << (unsigned)(lat)
              << std::endl;
  }

  return 1;
}
