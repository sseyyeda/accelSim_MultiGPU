#include "hip/hip_runtime.h"
// This code is a modification of L1 cache benchmark from
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking":
// https://arxiv.org/pdf/1804.06826.pdf

// This benchmark measures the maximum read bandwidth of L1 cache for 32 bit

#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "../../../hw_def/hw_def.h"

#define REPEAT_TIMES 256
// array size is half the L1 size (2) * float size (4)
#define ARRAY_SIZE (L1_SIZE / 8)

__global__ void l1_bw(uint64_t *startClk, uint64_t *stopClk, float *dsink,
                      float *posArray) {

  // thread index
  uint32_t tid = threadIdx.x;
  uint32_t uid = blockIdx.x * blockDim.x + tid;

  // a register to avoid compiler optimization
  float sink0 = 0;
  float sink1 = 0;
  float sink2 = 0;
  float sink3 = 0;

  // warp up L1 cache
  for (uint32_t i = tid * 4; i < ARRAY_SIZE; i += blockDim.x * 4) {
    float *ptr = posArray + i;
    // use ca modifier to cache the load in L1
    asm volatile("{\t\n"
                 ".reg .f32 data<4>;\n\t"
                 "ld.global.ca.v4.f32 {data0,data1,data2,data3}, [%4];\n\t"
                 "add.f32 %0, data0, %0;\n\t"
                 "add.f32 %1, data1, %1;\n\t"
                 "add.f32 %2, data2, %2;\n\t"
                 "add.f32 %3, data3, %3;\n\t"
                 "}"
                 : "+f"(sink0), "+f"(sink1), "+f"(sink2), "+f"(sink3)
                 : "l"(ptr)
                 : "memory");
  }

  // synchronize all threads
  asm volatile("bar.sync 0;");

  // start timing
  uint64_t start = 0;
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");

  // load data from l1 cache and accumulate
  for (uint32_t j = 0; j < REPEAT_TIMES; j++) {
    float *ptr = posArray + ((tid * 4 + (j * warpSize * 4)) % ARRAY_SIZE);
    asm volatile("{\t\n"
                 ".reg .f32 data<4>;\n\t"
                 "ld.global.ca.v4.f32 {data0,data1,data2,data3}, [%4];\n\t"
                 "add.f32 %0, data0, %0;\n\t"
                 "add.f32 %1, data1, %1;\n\t"
                 "add.f32 %2, data2, %2;\n\t"
                 "add.f32 %3, data3, %3;\n\t"
                 "}"
                 : "+f"(sink0), "+f"(sink1), "+f"(sink2), "+f"(sink3)
                 : "l"(ptr)
                 : "memory");
  }

  // synchronize all threads
  asm volatile("bar.sync 0;");

  // stop timing
  uint64_t stop = 0;
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");

  // write time and data back to memory
  startClk[uid] = start;
  stopClk[uid] = stop;
  dsink[uid] = sink0 + sink1 + sink2 + sink3;
}

int main() {
  intilizeDeviceProp(0);

  BLOCKS_NUM = 1;
  TOTAL_THREADS = THREADS_PER_BLOCK * BLOCKS_NUM;
  THREADS_PER_SM = THREADS_PER_BLOCK * BLOCKS_NUM;

  // ARRAY_SIZE has to be less than L1_SIZE
  assert(ARRAY_SIZE * sizeof(float) < L1_SIZE);

  uint64_t *startClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
  uint64_t *stopClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
  float *posArray = (float *)malloc(ARRAY_SIZE * sizeof(float));
  float *dsink = (float *)malloc(TOTAL_THREADS * sizeof(float));

  uint64_t *startClk_g;
  uint64_t *stopClk_g;
  float *posArray_g;
  float *dsink_g;

  for (uint32_t i = 0; i < ARRAY_SIZE; i++)
    posArray[i] = (float)i;

  gpuErrchk(hipMalloc(&startClk_g, TOTAL_THREADS * sizeof(uint64_t)));
  gpuErrchk(hipMalloc(&stopClk_g, TOTAL_THREADS * sizeof(uint64_t)));
  gpuErrchk(hipMalloc(&posArray_g, ARRAY_SIZE * sizeof(float)));
  gpuErrchk(hipMalloc(&dsink_g, TOTAL_THREADS * sizeof(float)));

  l1_bw<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, dsink_g,
                                           posArray_g);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchk(hipMemcpy(startClk, startClk_g, TOTAL_THREADS * sizeof(uint64_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS * sizeof(uint64_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(dsink, dsink_g, TOTAL_THREADS * sizeof(float),
                       hipMemcpyDeviceToHost));

  double bw, BW;
  uint64_t total_time =
      *std::max_element(&stopClk[0], &stopClk[TOTAL_THREADS]) -
      *std::min_element(&startClk[0], &startClk[TOTAL_THREADS]);
  bw = (double)(REPEAT_TIMES * THREADS_PER_SM * sizeof(float) * 4) /
       ((double)total_time);
  BW = bw * CLK_FREQUENCY * 1000000 / 1024 / 1024 / 1024;
  std::cout << "L1 bandwidth = " << bw << "(byte/clk/SM), " << BW
            << "(GB/s/SM)\n";
  std::cout << "Total Clk number = " << total_time << "\n";

  return 1;
}
