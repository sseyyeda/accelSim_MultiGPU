#include "hip/hip_runtime.h"
// Is L1 sector?

#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define L1_SIZE_FLOAT L1_SIZE / 4
// allocate large array size, larger than L1 size
#define ARRAY_SIZE L1_SIZE_FLOAT * 2
// we know the sector size is 8 floats (32B) from the l1_access_grain ubench
#define SECTOR_SIZE 8

#include "../../../hw_def/hw_def.h"

__global__ void l1_sector(uint32_t *startClk, uint32_t *stopClk, float *dsink,
                          float *posArray) {

  // thread index
  uint32_t tid = threadIdx.x;
  uint32_t uid = blockIdx.x * blockDim.x + tid;

  // a register to avoid compiler optimization
  float sink0 = 0;

  // populate l1 cache to warm up
  for (uint32_t i = tid; i < L1_SIZE_FLOAT; i += blockDim.x) {
    float *ptr = posArray + i;
    // use ca modifier to cache the load in L1
    asm volatile("{\t\n"
                 ".reg .f32 data;\n\t"
                 "ld.global.ca.f32 data, [%1];\n\t"
                 "add.f32 %0, data, %0;\n\t"
                 "}"
                 : "+f"(sink0)
                 : "l"(ptr)
                 : "memory");
  }

  // synchronize all threads
  asm volatile("bar.sync 0;");

  // kicks out one of the cache line and read a sector
  if (uid == 0) {
    sink0 += posArray[L1_SIZE_FLOAT + 1];
  }

  asm volatile("bar.sync 0;");

  uint32_t start = 0;
  uint32_t stop = 0;

  // start timing
  asm volatile("mov.u32 %0, %%clock;" : "=r"(start)::"memory");

  // load data from l1 cache and accumulate
  float *ptr = posArray + tid * SECTOR_SIZE;
  asm volatile("{\t\n"
               ".reg .f32 data;\n\t"
               "ld.global.ca.f32 data, [%1];\n\t"
               "add.f32 %0, data, %0;\n\t"
               "}"
               : "+f"(sink0)
               : "l"(ptr)
               : "memory");

  // stop timing
  asm volatile("mov.u32 %0, %%clock;" : "=r"(stop)::"memory");

  // synchronize all threads
  asm volatile("bar.sync 0;");

  // write time and data back to memory
  startClk[uid] = start;
  stopClk[uid] = stop;
  dsink[uid] = sink0;
}

int main() {
  intilizeDeviceProp(0);

  BLOCKS_NUM = 1;
  TOTAL_THREADS = THREADS_PER_BLOCK * BLOCKS_NUM;
  THREADS_PER_SM = THREADS_PER_BLOCK * BLOCKS_NUM;

  uint32_t *startClk = (uint32_t *)malloc(TOTAL_THREADS * sizeof(uint32_t));
  uint32_t *stopClk = (uint32_t *)malloc(TOTAL_THREADS * sizeof(uint32_t));
  float *posArray = (float *)malloc(ARRAY_SIZE * sizeof(float));
  float *dsink = (float *)malloc(TOTAL_THREADS * sizeof(float));

  uint32_t *startClk_g;
  uint32_t *stopClk_g;
  float *posArray_g;
  float *dsink_g;

  for (uint32_t i = 0; i < ARRAY_SIZE; i++)
    posArray[i] = (float)i;

  gpuErrchk(hipMalloc(&startClk_g, TOTAL_THREADS * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&stopClk_g, TOTAL_THREADS * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&posArray_g, ARRAY_SIZE * sizeof(float)));
  gpuErrchk(hipMalloc(&dsink_g, TOTAL_THREADS * sizeof(float)));

  gpuErrchk(hipMemcpy(posArray_g, posArray, ARRAY_SIZE * sizeof(float),
                       hipMemcpyHostToDevice));

  std::cout << "Launching L1 sector ubench" << std::endl;

  l1_sector<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, dsink_g,
                                               posArray_g);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchk(hipMemcpy(startClk, startClk_g, TOTAL_THREADS * sizeof(uint32_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS * sizeof(uint32_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(dsink, dsink_g, TOTAL_THREADS * sizeof(float),
                       hipMemcpyDeviceToHost));

  ofstream myfile;
  myfile.open("data.csv");
  myfile << "sectror_id, lat" << endl;
  for (unsigned i = 0; i < TOTAL_THREADS; i++) {
    myfile << i << "," << stopClk[i] - startClk[i] << endl;
  }

  std::cout << "Saving L1 sector data at data.csv" << std::endl;

  myfile.close();

  return 1;
}
